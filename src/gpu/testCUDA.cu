//
// Created by moura on 30/12/2022.
//


#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result != hipSuccess) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
                  file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__ void add(int a, int b, int* c) {
    *c = a + b;
}

int main() {
    int count;
    checkCudaErrors(hipGetDeviceCount(&count));

    cout << "Found " << count << " device(s)" << endl;

    for (int i = 0; i < count; ++i) {
        hipDeviceProp_t prop;

        checkCudaErrors(hipGetDeviceProperties(&prop, i));
        cout << "Device name: " << prop.name << endl;
        cout << "Total Memory: " << prop.totalGlobalMem / 1024.0 / 1024.0 << "MB" << endl;
        cout << "Max Threads per Block: " << prop.maxThreadsPerBlock << endl;
        cout << "Compute capability: " << prop.major << "." << prop.minor << endl;
    }

    int result;
    int* devResult;

    checkCudaErrors(hipMalloc((void**)&devResult, sizeof(int)));

    add<<<1, 1>>>(7, 8, devResult);
    //checkCudaErrors(cudaGetLastError());

    checkCudaErrors(hipMemcpy(&result, devResult, sizeof(int), hipMemcpyDeviceToHost));

    cout << "7 + 8 = " << result << endl;

    hipFree(devResult);

    return 0;
}